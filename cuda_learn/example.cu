#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "./common.cpp"

__global__ void add_kernel (float* a, float* b) {
    int i = threadIdx.x;
    b[i] = a[i] + 1.0f;
}

int main (void) {
    const int SIZE = 8;
    const float a[SIZE] = {1, 2, 3, 4, 5, 6, 7, 8};
    float b[SIZE] = {0};

	printf("a = {%f,%f,%f,%f,%f,%f,%f,%f}\n", a[0], a[1], a[2], a[3], a[4], a[5], a[6], a[7]);
    fflush (stdout);

    float * dev_a = nullptr;
    float * dev_b = nullptr;
    hipMalloc ((void**)&dev_a, SIZE * sizeof(float));
    hipMalloc ((void**)&dev_b, SIZE * sizeof(float));

    hipMemcpy (dev_a, a, SIZE * sizeof(float), hipMemcpyHostToDevice);

    add_kernel <<<1,SIZE>>> (dev_a, dev_b);
    hipDeviceSynchronize ();

    hipMemcpy (b, dev_b, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    printf("b = {%f,%f,%f,%f,%f,%f,%f,%f}\n", b[0], b[1], b[2], b[3], b[4], b[5], b[6], b[7]);
	fflush( stdout );
	// free device memory
	hipFree( dev_a );
	hipFree( dev_b );
	// done
	return 0;
}
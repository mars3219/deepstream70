
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello( void ) {
	printf( "hello CUDA!\n" );
}

int main( void ) {
	hello<<<1,1>>>();
	fflush( stdout );
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// main program for the CPU: compiled by MS-VC++
int main(void) {
	// host-side data
	const int SIZE = 8;
	const float a[SIZE] = { 1., 2., 3., 4., 5., 6., 7., 8. };
	float b[SIZE] = { 0., 0., 0., 0., 0., 0., 0., 0. };
	// print source
	printf("a = {%f,%f,%f,%f,%f,%f,%f,%f}\n", a[0], a[1], a[2], a[3], a[4], a[5], a[6], a[7]);
	fflush( stdout );
	// device-side data
	float* dev_a = nullptr;
	float* dev_b = nullptr;
	// allocate device memory
	hipMalloc( (void**)&dev_a, SIZE * sizeof(float) );
	hipMalloc( (void**)&dev_b, SIZE * sizeof(float) );
	// 3 copies
	hipMemcpy( dev_a, a, SIZE * sizeof(float), hipMemcpyHostToDevice); // dev_a = a;
	hipMemcpy( dev_b, dev_a, SIZE * sizeof(float), hipMemcpyDeviceToDevice); // dev_b = dev_a;
	hipMemcpy( b, dev_b, SIZE * sizeof(float), hipMemcpyDeviceToHost); // b = dev_b;
	// free device memory
	hipFree( dev_a );
	hipFree( dev_b );
	// print the result
	printf("b = {%f,%f,%f,%f,%f,%f,%f,%f}\n", b[0], b[1], b[2], b[3], b[4], b[5], b[6], b[7]);
	fflush( stdout );
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */

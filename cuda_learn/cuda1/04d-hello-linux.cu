
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello( void ) {
	printf( "hello CUDA %d !\n", threadIdx.x );
}

int main( void ) {
	hello<<<1,8>>>();
#if defined(__linux__)
	hipDeviceSynchronize();
#endif
	fflush( stdout );
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */

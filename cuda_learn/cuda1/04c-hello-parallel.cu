
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello( void ) {
	printf( "hello CUDA %d !\n", threadIdx.x );
}

int main( void ) {
	hello<<<8,2>>>();
	fflush( stdout );
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */

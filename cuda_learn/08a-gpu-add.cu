#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// kernel program for the device (GPU): compiled by NVCC
__global__ void add_kernel( int* c, const int* a, const int* b ) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


// main program for the CPU: compiled by MS-VC++
int main(void) {
	// host-side data
	const int SIZE = 5;
	const int a[SIZE] = { 1, 2, 3, 4, 5 };
	const int b[SIZE] = { 10, 20, 30, 40, 50 };
	int c[SIZE] = { 0 };
	// device-side data
	int* dev_a = nullptr;
	int* dev_b = nullptr;
	int* dev_c = nullptr;
	// allocate device memory
	hipMalloc( (void**)&dev_a, SIZE * sizeof(int) );
	hipMalloc( (void**)&dev_b, SIZE * sizeof(int) );
	hipMalloc( (void**)&dev_c, SIZE * sizeof(int) );
	// copy from host to device
	hipMemcpy( dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice ); // dev_a = a;
	hipMemcpy( dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice ); // dev_b = b;
	// launch a kernel on the GPU with one thread for each element.
	add_kernel<<<1,SIZE>>>( dev_c, dev_a, dev_b );		// dev_c = dev_a + dev_b;
	hipDeviceSynchronize();
	hipError_t err = hipPeekAtLastError();
	if (hipSuccess != err) {
		printf("CUDA: ERROR: cuda failure \"%s\"\n", hipGetErrorString(err));
		exit(1);
	} else {
		printf("CUDA: success\n");
	}
	// copy from device to host
	hipMemcpy( c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost ); // c = dev_c;
	// free device memory
	hipFree(dev_a );
	hipFree(dev_b );
	hipFree(dev_c );
	// print the result
	printf("{%d,%d,%d,%d,%d} + {%d,%d,%d,%d,%d} = {%d,%d,%d,%d,%d}\n",
	       a[0], a[1], a[2], a[3], a[4],
	       b[0], b[1], b[2], b[3], b[4],
	       c[0], c[1], c[2], c[3], c[4]);
	// done
	fflush( stdout );
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
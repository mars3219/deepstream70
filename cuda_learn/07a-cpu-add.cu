#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// main program for the CPU: compiled by MS-VC++
int main(void) {
	// host-side data
	const int SIZE = 5;
	const int a[SIZE] = { 1, 2, 3, 4, 5 };
	const int b[SIZE] = { 10, 20, 30, 40, 50 };
	int c[SIZE] = { 0 };
	// calculate the addition
	for (register int i = 0; i < SIZE; ++i) {
		c[i] = a[i] + b[i];
	}
	// print the result
	printf("{%d,%d,%d,%d,%d} + {%d,%d,%d,%d,%d} = {%d,%d,%d,%d,%d}\n",
	       a[0], a[1], a[2], a[3], a[4],
	       b[0], b[1], b[2], b[3], b[4],
	       c[0], c[1], c[2], c[3], c[4]);
	fflush( stdout );
	// done
	return 0;
}

/* (c) 2021-2022. biztripcru@gmail.com. All rights reserved. */
